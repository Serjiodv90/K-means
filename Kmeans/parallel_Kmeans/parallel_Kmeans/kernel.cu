#include "hip/hip_runtime.h"

#include "kernel.h"

//better to access from a single process to avoid unknown behavior. so access only from master

hipError_t calculateNewPointPositionViaTime_Cuda(Point::PointAsStruct* pointsArr, int numberOfPoints, double currentTime);

__global__ void calcNewPosition(Point::PointAsStruct* pointsArr, int numberOfPoints, double currentTime)
{
	int threadId = threadIdx.x;
	int blockId = blockIdx.x;
	int index = threadId + blockId * MAX_THREADS_FOR_CUDA;
	if (index < numberOfPoints)
	{
		pointsArr[index].current_x = pointsArr[index].X0 + (currentTime * pointsArr[index].velocity_x);
		pointsArr[index].current_y = pointsArr[index].Y0 + (currentTime * pointsArr[index].velocity_y);
		pointsArr[index].current_z = pointsArr[index].Z0 + (currentTime * pointsArr[index].velocity_z);
	}

}


// Helper function for using CUDA to add vectors in parallel.
hipError_t calculateNewPointPositionViaTime_Cuda(Point::PointAsStruct* pointsArr, int numberOfPoints, double currentTime)
{

	int numOfBlocksForCuda;
	Point::PointAsStruct* pointsArr_device;
    hipError_t cudaStatus;

	numOfBlocksForCuda = 1 + ((numberOfPoints - 1) / MAX_THREADS_FOR_CUDA);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		hipFree(pointsArr_device);
    }

    // Allocate GPU buffer for array of Points.
    cudaStatus = hipMalloc((void**)&pointsArr_device, numberOfPoints * sizeof(Point::PointAsStruct));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
		hipFree(pointsArr_device);
    }

    // Copy array of Points from host memory to GPU buffers.
    cudaStatus = hipMemcpy(pointsArr_device, pointsArr, numberOfPoints * sizeof(Point::PointAsStruct), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		hipFree(pointsArr_device);
    }

    // Launch a kernel on the GPU with one thread for each element.
    calcNewPosition<<<numOfBlocksForCuda, MAX_THREADS_FOR_CUDA>>>(pointsArr_device, numberOfPoints, currentTime);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		hipFree(pointsArr_device);
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		hipFree(pointsArr_device);
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(pointsArr, pointsArr_device, numberOfPoints * sizeof(Point::PointAsStruct), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		hipFree(pointsArr_device);
    }
    
    return cudaStatus;
}

